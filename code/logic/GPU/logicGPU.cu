#include "logicGPU.cuh"

/**
 * Compute SpMM with CSR matrix in CUDA parallel mode
 * @param matrix input CSR matrix
 * @param vector input vector
 * @param res_vec output vector
 * @param NZ number of NZ elements in the CSR matrix
 * @return usec of execution
 */
double parallel_cuda_csr(CSRMatrix *matrix, MultiVector *vector, MultiVector *res_vec, int NZ) {

    // Reset device
    checkCudaErrors(hipDeviceReset());

    // Initialize CUDA variables
    int *d_JA;
    int *d_IRP;
    int *d_row_blocks;
    double *d_AS;
    double *d_val_vec;
    double *d_val_res;

    // Initialize matrix parameters
    int M = matrix->M;
    int *JA = matrix->JA;
    int *IRP = matrix->IRP;
    double *AS = matrix->AS;

    // Initialize vector parameters
    int m = vector->m;
    int n = vector->n;
    double *val = vector->val;

    // Compute row blocks
    int block_count;
    int *row_blocks = find_row_blocks(M, IRP, &block_count);

    // Set block and grid sizes
    const dim3 block_size = dim3(BLOCK_SIZE_CSR);
    const dim3 grid_size = dim3(block_count);

    // Allocate memory on the device
    checkCudaErrors(hipMalloc((void **) &d_IRP, (M + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_JA, NZ * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, NZ * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_val_vec, m * n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_val_res, m * n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_row_blocks, (block_count + 1) * sizeof(int)));

    // Send data to the device
    checkCudaErrors(hipMemcpy(d_IRP, IRP, (M + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_JA, JA, NZ * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, AS, NZ * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_val_vec, val, m * n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_row_blocks, row_blocks, (block_count + 1) * sizeof(int), hipMemcpyHostToDevice));

    // Set to 0 all values of the result multivector
    checkCudaErrors(hipMemset(d_val_res, 0, m * n * sizeof(double)));

    // Generate the timer
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Start the timer
    checkCudaErrors(hipEventRecord(start, nullptr));

    // Run CSR CUDA kernel
    csr_adaptive<<<grid_size, block_size>>>(n, d_JA, d_IRP, d_AS, d_val_vec, d_val_res, d_row_blocks);

    // Wait for the kernel to complete
    checkCudaErrors(hipDeviceSynchronize());

    // Stop the timer
    checkCudaErrors(hipEventRecord(stop, nullptr));
    checkCudaErrors(hipEventSynchronize(stop));

    // Compute elapsed time
    float time;
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));

    // Destroy the timer
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    // Allocate result values vector and set res_vec parameters
    auto *val_res = (double *) calloc(M * n, sizeof(double));
    checkCudaErrors(hipMemcpy(val_res, d_val_res, M * n * sizeof(double), hipMemcpyDeviceToHost));
    res_vec->m = m;
    res_vec->n = n;
    res_vec->val = val_res;

    // Free memory on the device
    checkCudaErrors(hipFree(d_IRP));
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_val_vec));
    checkCudaErrors(hipFree(d_val_res));
    checkCudaErrors(hipFree(d_row_blocks));

    // Free row blocks
    free(row_blocks);

    // Return elapsed time
    return (double) time;
}

/**
 * Compute SpMM with ELLPACK matrix in CUDA parallel mode
 * @param matrix input ELLPACK matrix
 * @param vector input vector
 * @param res_vec output vector
 * @return usec of executions
 */
double parallel_cuda_ell(ELLMatrix *matrix, MultiVector *vector, MultiVector *res_vec) {

    // Reset device
    checkCudaErrors(hipDeviceReset());

    // Initialize CUDA variables
    int *d_JA;
    double *d_AS;
    double *d_val_vec;
    double *d_val_res;

    // Initialize matrix parameters
    int M = matrix->M;
    int MAXNZ = matrix->MAXNZ;
    int *JA = matrix->JA;
    double *AS = matrix->AS;

    // Initialize vector parameters
    int m = vector->m;
    int n = vector->n;
    double *val = vector->val;

    // Allocate result values vector
    auto *val_res = (double *) calloc(m * n, sizeof(double));

    // Set block and grid sizes
    const dim3 block_size = dim3(BLOCK_SIZE_ELLPACK);
    const dim3 grid_size = dim3(M + 1);

    // Allocate memory on the device
    checkCudaErrors(hipMalloc((void **) &d_JA, MAXNZ * M * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, MAXNZ * M * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_val_vec, m * n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_val_res, m * n * sizeof(double)));

    // Send data to the device
    checkCudaErrors(hipMemcpy(d_JA, JA, MAXNZ * M * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, AS, MAXNZ * M * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_val_vec, val, m * n * sizeof(double), hipMemcpyHostToDevice));

    // Generate the timer
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Start the timer
    hipEventRecord(start, nullptr);

    // Run ELLPACK CUDA kernel
    ell_kernel<<<grid_size, block_size>>>(M, n, MAXNZ, d_JA, d_AS, d_val_vec, d_val_res);

    // Wait for the kernel to complete
    checkCudaErrors(hipDeviceSynchronize());

    // Stop the timer
    hipEventRecord(stop, nullptr);
    checkCudaErrors(hipEventSynchronize(stop));

    // Compute elapsed time
    float time;
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));

    // Destroy the timer
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    // Set res_vec parameters
    checkCudaErrors(hipMemcpy(val_res, d_val_res, m * n * sizeof(double), hipMemcpyDeviceToHost));
    res_vec->m = m;
    res_vec->n = n;
    res_vec->val = val_res;

    // Free memory on the device
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));
    checkCudaErrors(hipFree(d_val_vec));
    checkCudaErrors(hipFree(d_val_res));

    // Return elapsed time
    return (double) time;
}